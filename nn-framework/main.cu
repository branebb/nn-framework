// Standard library
#include <iostream>
#include <stdio.h>
#include <time.h>

// Nvidia
#include <hip/driver_types.h>
#include <hip/hip_runtime.h>

// Project headers
#include "nn-framework/utils/error_check_cuda.hpp"
#include "nn-framework/headers/structures/neural_network.hh"
#include "nn-framework/headers/cost_functions/MSEcost.hh"
#include "nn-framework/headers/layers/linear_layer.hh"
#include "nn-framework/headers/layers/tanh_activation.hh"
#include "nn-framework/headers/layers/relu_activation.hh"
#include "nn-framework/headers/layers/sigmoid_activation.hh"
#include "nn-framework/headers/structures/matrix.hh"
#include "nn-framework/coordinates_test.hh"
#include "nn-framework/headers/layers/softmax_activation.hh"


int main()
{
    srand(time(NULL));

    MSECost MSE;
    NeuralNetwork nn(&MSE);

    nn.addLayer(new LinearLayer("linear1", Dimensions(2, 10)));
    nn.addLayer(new ReLUActivation("softmaxtest"));
    nn.addLayer(new LinearLayer("linear2", Dimensions(10, 2)));
    nn.addLayer(new SoftmaxActivation("sigmoid"));

    CoordinatesDataset dataset(100, 20);

    Matrix Y;
    for (int epoch = 0; epoch < 1001; epoch++) 
    {
        float cost = 0.0;

        for (int batch = 0; batch < dataset.getNumOfBatches() - 1; batch++) 
        {
            Y = nn.forward(dataset.getBatches().at(batch));
            nn.backprop(Y, dataset.getTargets().at(batch));
            cost += MSE.cost(Y, dataset.getTargets().at(batch));
        }

        if (epoch % 100 == 0) 
        {
            std::cout << "Epoch: " << epoch
                << ", Cost: " << cost / dataset.getNumOfBatches()
                << std::endl;
        }
    }

    cuda_check(hipDeviceSynchronize());

    return 0;
}