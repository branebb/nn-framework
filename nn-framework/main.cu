// Standard library
#include <iostream>
#include <stdio.h>
#include <time.h>

// Nvidia
#include <hip/driver_types.h>
#include <hip/hip_runtime.h>

// Project headers
#include "nn-framework/utils/error_check_cuda.hpp"
#include "nn-framework/headers/structures/neural_network.hh"
#include "nn-framework/headers/cost_functions/MSEcost.hh"
#include "nn-framework/headers/layers/linear_layer.hh"
#include "nn-framework/headers/layers/tanh_activation.hh"
#include "nn-framework/headers/layers/relu_activation.hh"
#include "nn-framework/headers/layers/sigmoid_activation.hh"
#include "nn-framework/headers/structures/matrix.hh"
#include "nn-framework/coordinates_test.hh"
#include "nn-framework/headers/layers/softmax_activation.hh"

int main()
{
    MSECost MSE;
    float lr = 0.1;
    NeuralNetwork nn(&MSE, lr);

    nn.addLayer(new LinearLayer("linear1", Dimensions(2, 30)));
    nn.addLayer(new ReLUActivation("softmaxtest"));
    nn.addLayer(new LinearLayer("linear2", Dimensions(30, 2)));
    nn.addLayer(new SoftmaxActivation("softmax"));

    CoordinatesDataset dataset(100, 20);

    Matrix Y;

    for (int epoch = 0; epoch < 101; epoch++) 
    {
        float cost = 0.0;

        for (int batch = 0; batch < dataset.getNumOfBatches() - 1; batch++) 
        {
            Y = nn.forward(dataset.getBatches().at(batch));
            nn.backprop(Y, dataset.getTargets().at(batch));
            cost += MSE.cost(Y, dataset.getTargets().at(batch));
        }

        if (epoch % 10 == 0) 
                std::cout << "Epoch: " << epoch << ", Cost: " << cost / dataset.getNumOfBatches() << std::endl;
    }

    Y = nn.forward(dataset.getBatches().at(dataset.getNumOfBatches() - 1));

    float accuracy = nn.computeAccuracy(Y, dataset.getTargets().at(dataset.getNumOfBatches() - 1));

    std::cout << "Accuracy: " << accuracy << std::endl;

    return 0;
}