// Standard library
#include <iostream>
#include <stdio.h>

// Nvidia
#include <hip/driver_types.h>
#include <hip/hip_runtime.h>

// Project headers
#include "nn-framework/utils/error_check_cuda.hpp"
#include "nn-framework/headers/structures/neural_network.hh"
#include "nn-framework/headers/cost_functions/MSEcost.hh"
#include "nn-framework/headers/layers/linear_layer.hh"
#include "nn-framework/headers/layers/tanh_activation.hh"
#include "nn-framework/headers/layers/relu_activation.hh"
#include "nn-framework/headers/layers/sigmoid_activation.hh"
#include "nn-framework/headers/structures/matrix.hh"
#include "nn-framework/coordinates_test.hh"
#include "nn-framework/headers/layers/softmax_activation.hh"
#include "nn-framework/headers/optimizers/optimizer.hh"
#include "nn-framework/headers/optimizers/gradient.hh"
#include "nn-framework/headers/optimizers/adam.hh"
#include "nn-framework/headers/regularization/L2.hh"
#include "nn-framework/mnist/mnist.hh"
#include "nn-framework/headers/cost_functions/CrossEntropy.hh"

int main()
{
    float lambda = 0.00f;

    float beta1 = 0.9f;
    float beta2 = 0.999f;
    float epsilon = 1e-8f;

    float learning_rate = 0.01f;

    MNIST traindata(32, 1875, "datasets/mnist_train.csv");

    L2 L2(lambda);

    CrossEntropyCost crossEntropy(&L2);

    AdamOptimizer adam(beta1, beta2, epsilon);

    MSECost MSE(&L2);

    NeuralNetwork nn(&crossEntropy, &adam, &L2, learning_rate);

    nn.addLayer(new LinearLayer("linear1", Dimensions(784, 128)));
    nn.addLayer(new ReLUActivation("relu"));
    nn.addLayer(new LinearLayer("linear2", Dimensions(128, 64)));
    nn.addLayer(new ReLUActivation("relu2"));
    nn.addLayer(new LinearLayer("linear3", Dimensions(64, 10)));
    nn.addLayer(new SoftmaxActivation("softmax"));

    Matrix Y;

    for (int epoch = 0; epoch < 11; epoch++) 
    {
        float cost = 0.0;

        for (int batch = 0; batch < traindata.getNumOfBatches(); batch++) 
        {
            Y = nn.forward(traindata.getBatches().at(batch));
            nn.backprop(Y, traindata.getTargets().at(batch));
        
            LinearLayer* linearLayer = dynamic_cast<LinearLayer*>(nn.getLayers()[2]);
            Matrix layerW = linearLayer->getWeightsMatrix();
            cost += crossEntropy.cost(Y, traindata.getTargets().at(batch), layerW);
        }

        if (epoch % 1 == 0) 
                std::cout << "Epoch: " << epoch << ", Cost: " << cost / traindata.getNumOfBatches() << std::endl;
    }

    MNIST testdata(32, 312, "datasets/mnist_test.csv");
    Matrix A;

    float accuracy = 0.0f;
    for (int i = 0; i < testdata.getNumOfBatches(); i++)
    {
        A = nn.forward(testdata.getBatches().at(i));
        accuracy += nn.computeAccuracy(A, testdata.getTargets().at(i));
    }
    
    std::cout << "Accuracy on test data: " << accuracy / testdata.getNumOfBatches();

    return 0;
}