// Standard library
#include <iostream>
#include <stdio.h>
#include <time.h>

// Nvidia
#include <hip/driver_types.h>
#include <hip/hip_runtime.h>

// Project headers
#include "nn-framework/utils/error_check_cuda.hpp"
#include "nn-framework/headers/structures/neural_network.hh"
#include "nn-framework/headers/cost_functions/MSEcost.hh"
#include "nn-framework/headers/layers/linear_layer.hh"
#include "nn-framework/headers/layers/tanh_activation.hh"
#include "nn-framework/headers/layers/relu_activation.hh"
#include "nn-framework/headers/layers/sigmoid_activation.hh"
#include "nn-framework/headers/structures/matrix.hh"
#include "nn-framework/coordinates_test.hh"
#include "nn-framework/headers/layers/softmax_activation.hh"
#include "nn-framework/headers/optimizers/optimizer.hh"
#include "nn-framework/headers/optimizers/gradient.hh"
#include "nn-framework/headers/optimizers/adam.hh"
#include "nn-framework/headers/regularization/L2.hh"
#include "nn-framework/mnist.hh"

int main()
{
    MNIST traindata(32, 1875, "datasets/mnist_train.csv");

    float lambda = 0.01f;
    L2 l2(lambda);

    float beta1 = 0.9f;
    float beta2 = 0.999f;
    float epsilon = 1e-4f;

    AdamOptimizer adam(beta1, beta2, epsilon);

    MSECost MSE(&l2);

    Gradient grad;

    float lr = 0.001f;

    NeuralNetwork nn(&MSE, &adam, &l2, lr);

    nn.addLayer(new LinearLayer("linear1", Dimensions(784, 128)));
    nn.addLayer(new ReLUActivation("relu"));
    nn.addLayer(new LinearLayer("linear2", Dimensions(128, 64)));
    nn.addLayer(new ReLUActivation("relu"));
    nn.addLayer(new LinearLayer("linear2", Dimensions(64, 10)));
    nn.addLayer(new SoftmaxActivation("softmax"));

    Matrix Y;

    for (int epoch = 0; epoch < 11; epoch++) 
    {
        float cost = 0.0;

        for (int batch = 0; batch < traindata.getNumOfBatches(); batch++) 
        {
            Y = nn.forward(traindata.getBatches().at(batch));
            nn.backprop(Y, traindata.getTargets().at(batch));
        
            LinearLayer* linearLayer = dynamic_cast<LinearLayer*>(nn.getLayers()[2]);
            Matrix layerW = linearLayer->getWeightsMatrix();
            cost += MSE.cost(Y, traindata.getTargets().at(batch), layerW);
        }

        if (epoch % 1 == 0) 
                std::cout << "Epoch: " << epoch << ", Cost: " << cost / traindata.getNumOfBatches() << std::endl;
    }

    MNIST testdata(32, 312, "datasets/mnist_test.csv");
    Matrix A;

    float accuracy = 0.0f;
    for (int i = 0; i < testdata.getNumOfBatches(); i++)
    {
        A = nn.forward(testdata.getBatches().at(i));
        accuracy += nn.computeAccuracy(A, testdata.getTargets().at(i));
    }
    
    std::cout << "Accuracy on test data: " << accuracy / testdata.getNumOfBatches();

    // Epoch: 0, Cost: 0.273616
    // Epoch: 1, Cost: 0.243005
    // Epoch: 2, Cost: 0.222543
    // Epoch: 3, Cost: 0.207329
    // Epoch: 4, Cost: 0.18212
    // Epoch: 5, Cost: 0.176571
    // Epoch: 6, Cost: 0.153203
    // Epoch: 7, Cost: 0.149846
    // Epoch: 8, Cost: 0.140936
    // Epoch: 9, Cost: 0.134518
    // Epoch: 10, Cost: 0.129513
    // Accuracy on test data: 0.88111
    // Last run on this network with MNIST digits

    return 0;

}