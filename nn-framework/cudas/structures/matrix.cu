#include "hip/hip_runtime.h"
#include "nn-framework/headers/structures/matrix.hh"
#include "nn-framework/utils/error_check_cuda.hpp"

Matrix::Matrix(size_t x_dim, size_t y_dim) :
	dims(x_dim, y_dim), 
    deviceData(nullptr), 
    hostData(nullptr),
	isDeviceAllocated(false), 
    isHostAllocated(false)
{ }

Matrix::Matrix(Dimensions dims) :
	Matrix(dims.x, dims.y)
{ }

void Matrix::allocateCudaMemory() 
{
	if (!isDeviceAllocated) 
    {
		float* device_memory = nullptr;
		hipMalloc(&device_memory, dims.x * dims.y * sizeof(float));
		// cuda_check(hipDeviceSynchronize());
		deviceData = std::shared_ptr<float>(device_memory, [&](float* ptr) { hipFree(ptr); });
		isDeviceAllocated = true;
	}
}

void Matrix::allocateHostMemory() 
{
	if (!isHostAllocated) 
    {
		hostData = std::shared_ptr<float>(new float[dims.x * dims.y], [&](float* ptr) { delete[] ptr; });
		isHostAllocated = true;
	}
}

void Matrix::allocateMemory() 
{
	allocateCudaMemory();
	allocateHostMemory();
}

void Matrix::allocateMemoryIfNotAllocated(Dimensions dims) 
{
	if (!isDeviceAllocated && !isHostAllocated) 
    {
		this->dims = dims;
		allocateMemory();
	}
}

void Matrix::copyHostToDevice() 
{
	if (isDeviceAllocated && isHostAllocated) 
    {
		hipMemcpy(deviceData.get(), hostData.get(), dims.x * dims.y * sizeof(float), hipMemcpyHostToDevice);
		// cuda_check(hipDeviceSynchronize());
	}
}

void Matrix::copyDeviceToHost() 
{
	if (isDeviceAllocated && isHostAllocated) 
    {
		hipMemcpy(hostData.get(), deviceData.get(), dims.x * dims.y * sizeof(float), hipMemcpyDeviceToHost);
		// cuda_check(hipDeviceSynchronize());
	}
}

float& Matrix::operator[](const int index) 
{
	return hostData.get()[index];
}

const float& Matrix::operator[](const int index) const 
{
	return hostData.get()[index];
}

void Matrix::oneHotEncoding()
{
    for(int col = 0; col < dims.x; col++)
	{
		float max = -1.0f;
		int maxInd = -1;

		for(int row = 0; row < dims.y; row++)
		{
			float current = hostData.get()[col + dims.x * row];
			if(current > max)
			{
				max = current;
				maxInd = col + dims.x * row;
			}
		}

		for(int row = 0; row < dims.y; row++)
		{
			if(col + dims.x * row == maxInd)
				hostData.get()[col + dims.x * row] = 1.0f;
			else
				hostData.get()[col + dims.x * row] = 0.0f;
		}
	}
}

bool Matrix::deviceAllocation() { return this->isDeviceAllocated; }