#include "hip/hip_runtime.h"
#include "nn-framework/headers/layers/linear_layer.hh"
#include "nn-framework/utils/error_check_cuda.hpp"

#include <string>
#include <random>
#include <assert.h>

__global__ void linearLayerForward(float *W, float *A, float *Z, float *b, int W_x_dim, int W_y_dim, int A_x_dim, int A_y_dim)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int Z_x_dim = A_x_dim;
    int Z_y_dim = W_y_dim;

    float Z_value = 0;

    if (row < Z_y_dim && col < Z_x_dim)
    {
        for (int i = 0; i < W_x_dim; i++)
            Z_value += W[row * W_x_dim + i] * A[i * A_x_dim + col];

        Z[row * Z_x_dim + col] = Z_value + b[row];
    }
}

__global__ void linearLayerBackprop(float *W, float *dZ, float *dA, int W_x_dim, int W_y_dim, int dZ_x_dim, int dZ_y_dim)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // W.T
    int dA_x_dim = dZ_x_dim;
    int dA_y_dim = W_x_dim;

    float dA_value = 0.0f;

    if (row < dA_y_dim && col < dA_x_dim)
    {
        for (int i = 0; i < W_y_dim; i++)
            dA_value += W[i * W_x_dim + row] * dZ[i * dZ_x_dim + col];

        dA[row * dA_x_dim + col] = dA_value;
    }
}

__global__ void linearLayerCalculateWGradient(float *dZ, float *A, float *dW, int dZ_x_dim, int dZ_y_dim, int A_x_dim, int A_y_dim)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // A.T
    int W_x_dim = A_y_dim;
    int W_y_dim = dZ_y_dim;

    float dW_value = 0.0f;

    if (row < W_y_dim && col < W_x_dim)
    {
        for (int i = 0; i < dZ_x_dim; i++)
            dW_value += dZ[row * dZ_x_dim + i] * A[col * A_x_dim + i];

        dW[row * W_x_dim + col] = (dW_value / A_x_dim);
    }
}

__global__ void linearLayerCalculateBGradient(float *dZ, float *db, int dZ_x_dim, int dZ_y_dim, int b_x_dim)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col < dZ_x_dim)
    {
        float dB_value = 0.0f;
        
        for (int row = 0; row < dZ_y_dim; row++)
        {
            dB_value += dZ[row * dZ_x_dim + col];
        }
        
        db[col] = dB_value / dZ_x_dim;
    }
}

LinearLayer::LinearLayer(std::string name, Dimensions W_dims) : W(W_dims), b(W_dims.y, 1)
{
    this->name = name;
    b.allocateMemory();
    W.allocateMemory();
    initializeBiasWithZeros();
    initializeWeightsRandomly();
}

LinearLayer::~LinearLayer() {}

void LinearLayer::initializeWeightsRandomly()
{
    std::mt19937 rng(std::random_device{}());
    std::uniform_real_distribution<float> dist(-1.0f, 1.0f);

    for (int x = 0; x < W.dims.x; x++)
        for (int y = 0; y < W.dims.y; y++)
            W[y * W.dims.x + x] = dist(rng);

    W.copyHostToDevice();
}

void LinearLayer::initializeBiasWithZeros()
{
    for (int x = 0; x < b.dims.x; x++)
        b[x] = 0;

    b.copyHostToDevice();
}

Matrix &LinearLayer::forward(Matrix &A)
{
    // Check if W and A are chained
    assert(W.dims.x == A.dims.y);

    this->A = A;

    Dimensions Z_dims(A.dims.x, W.dims.y);

    Z.allocateMemoryIfNotAllocated(Z_dims);

    computeStoreLayerOutput(A);

    // cuda_check(hipDeviceSynchronize());

    return Z;
}

void LinearLayer::computeStoreLayerOutput(Matrix &A)
{
    dim3 block_size(32, 32);
    dim3 num_of_blocks((Z.dims.x + block_size.x - 1) / block_size.x, (Z.dims.y + block_size.y - 1) / block_size.y);

    linearLayerForward<<<num_of_blocks, block_size>>>(W.deviceData.get(), A.deviceData.get(), Z.deviceData.get(), b.deviceData.get(), W.dims.x, W.dims.y, A.dims.x, A.dims.y);

    cuda_check(hipDeviceSynchronize());
}

Matrix &LinearLayer::backprop(Matrix &dZ, float learning_rate)
{
    dA.allocateMemoryIfNotAllocated(A.dims);

    computeStoreBackpropError(dZ);

    dW.allocateMemoryIfNotAllocated(W.dims);
    db.allocateMemoryIfNotAllocated(b.dims);

    computeStoreWGradient(dZ);

    computeStoreBGradient(dZ);

    if(regularization)
    {
        regularization->gradientRegularization(W, dW, W.dims.x * W.dims.y);
    }
    
    optimizer->updateStep(dW, W, db, b, learning_rate);
    
    // cuda_check(hipDeviceSynchronize());

    return dA;
}

void LinearLayer::computeStoreBackpropError(Matrix &dZ)
{
    dim3 block_size(32, 32);
    dim3 num_of_blocks((A.dims.x + block_size.x - 1) / block_size.x, (A.dims.y + block_size.y - 1) / block_size.y);

    linearLayerBackprop<<<num_of_blocks, block_size>>>(W.deviceData.get(), dZ.deviceData.get(), dA.deviceData.get(), W.dims.x, W.dims.y, dZ.dims.x, dZ.dims.y);

    // cuda_check(hipDeviceSynchronize());
}

void LinearLayer::computeStoreWGradient(Matrix &dZ)
{
    dim3 block_size(32, 32);
    dim3 num_of_blocks((W.dims.x + block_size.x - 1) / block_size.x, (W.dims.y + block_size.y - 1) / block_size.y);

    linearLayerCalculateWGradient<<<num_of_blocks, block_size>>>(dZ.deviceData.get(), A.deviceData.get(), dW.deviceData.get(), dZ.dims.x, dZ.dims.y, A.dims.x, A.dims.y);

    // cuda_check(hipDeviceSynchronize());
}

void LinearLayer::computeStoreBGradient(Matrix &dZ)
{
    dim3 block_size(1024);
    dim3 num_of_blocks((dZ.dims.y * dZ.dims.x + block_size.x - 1) / block_size.x);

    linearLayerCalculateBGradient<<<num_of_blocks, block_size>>>(dZ.deviceData.get(), db.deviceData.get(), dZ.dims.x, dZ.dims.y, b.dims.x);

    // cuda_check(hipDeviceSynchronize());
}

int LinearLayer::getXDim() const { return W.dims.x; }

int LinearLayer::getYDim() const { return W.dims.y; }

Matrix LinearLayer::getWeightsMatrix() const { return W; }

Matrix LinearLayer::getBiasVector() const { return b; }

void LinearLayer::setOptimizer(Optimizer* optimizer) { this->optimizer = optimizer; }

void LinearLayer::setRegularization(Regularization* regularization) { this->regularization = regularization; }
